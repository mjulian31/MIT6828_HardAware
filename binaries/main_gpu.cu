#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern __global__ void _Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32(double* out, double* A, double* B, int dim);

int main(int argc, char *argv[]) {
	int dim = 1024;
	if (argc >= 2) {
		dim = atoi(argv[1]);
	}
	double* A = malloc(sizeof(double) * dim * dim);
	double* B = malloc(sizeof(double) * dim * dim);
	double* output = malloc(sizeof(double) * dim * dim);
  double *d_A, *d_B, *d_output;
	hipMalloc((void **)&A, sizeof(double) * dim * dim);
	hipMalloc((void **)&B, sizeof(double) * dim * dim);
	hipMalloc((void **)&output, sizeof(double) * dim * dim);
  hipMemcpy(d_A, A, dim*dim, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, dim*dim, hipMemcpyHostToDevice);
	_Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32<<<dim, dim>>>(output, A, B, dim);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, dim*dim, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_output);
	free(A);
	free(B);
	free(C);
}
