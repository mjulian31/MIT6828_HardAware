#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern __global__ void _Z38julia_coalesced_matmul_kernelNOT__393513CuDeviceArrayI7Float32Li2ELi1EES_IS0_Li2ELi1EES_IS0_Li2ELi1EE(double* out, double* A, double* B, int dim);

int main(int argc, char *argv[]) {
	int dim = 1024;
	if (argc >= 2) {
		dim = atoi(argv[1]);
	}
	double* A = malloc(sizeof(double) * dim * dim);
	double* B = malloc(sizeof(double) * dim * dim);
	double* output = malloc(sizeof(double) * dim * dim);
  double *d_A, *d_B, *d_output;
	hipMalloc((void **)&A, sizeof(double) * dim * dim);
	hipMalloc((void **)&B, sizeof(double) * dim * dim);
	hipMalloc((void **)&output, sizeof(double) * dim * dim);
  hipMemcpy(d_A, A, dim*dim, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, dim*dim, hipMemcpyHostToDevice);
	_Z38julia_coalesced_matmul_kernelNOT__393513CuDeviceArrayI7Float32Li2ELi1EES_IS0_Li2ELi1EES_IS0_Li2ELi1EE<<<dim, dim>>>(output, A, B, dim);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, dim*dim, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_output);
	free(A);
	free(B);
	free(C);
}
