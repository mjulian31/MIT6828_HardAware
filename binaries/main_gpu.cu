#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


extern "C" __device__ void _Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32(double* out, double* A, double* B, int dim);

__global__ void __matmul(double* out, double* A, double* B, int dim) {
	_Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32(out, A, B, dim);
}

int main(int argc, char *argv[]) {
	int dim = 1024;
	if (argc >= 2) {
		dim = atoi(argv[1]);
	}
	printf("dim: %i\n", dim);
	double* A = (double*)malloc(sizeof(double) * dim * dim);
	A[1] = 0xfffffff;;
	double* B = (double*)malloc(sizeof(double) * dim * dim);
	B[1] = 0xfffffff;
	double* output = (double*)malloc(sizeof(double) * dim * dim);
	printf("start, out1: %f, a1: %f, b1: %f\n", output[1], A[1], B[1]);
  double *d_A, *d_B, *d_output;
	hipMalloc((void **)&d_A, sizeof(double) * dim * dim);
	hipMalloc((void **)&d_B, sizeof(double) * dim * dim);
	hipMalloc((void **)&d_output, sizeof(double) * dim * dim);
  hipMemcpy(d_A, A, dim*dim, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, dim*dim, hipMemcpyHostToDevice);
	__matmul<<<dim, dim>>>(output, A, B, dim);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, dim*dim, hipMemcpyDeviceToHost);
  printf("done, out1: %f\n", output[1]);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_output);
	free(A);
	free(B);
	free(output);
}
