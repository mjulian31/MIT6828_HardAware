#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" __device__ void _Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32(double* out, double* A, double* B, int dim);

__global__ void __matmul(double* out, double* A, double* B, int dim) {
	_Z12julia_matmul3PtrI7Float64ES_IS0_ES_IS0_E5Int32(output, A, B, dim);
}

int main(int argc, char *argv[]) {
	int dim = 1024;
	if (argc >= 2) {
		dim = atoi(argv[1]);
	}
	double* A = (double*)malloc(sizeof(double) * dim * dim);
	double* B = (double*)malloc(sizeof(double) * dim * dim);
	double* output = (double*)malloc(sizeof(double) * dim * dim);
  double *d_A, *d_B, *d_output;
	hipMalloc((void **)&d_A, sizeof(double) * dim * dim);
	hipMalloc((void **)&d_B, sizeof(double) * dim * dim);
	hipMalloc((void **)&d_output, sizeof(double) * dim * dim);
  hipMemcpy(d_A, A, dim*dim, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, dim*dim, hipMemcpyHostToDevice);
	__matmul<<<dim, dim>>>(output, A, B, dim);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, dim*dim, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_output);
	free(A);
	free(B);
	free(output);
}
